
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define ull unsigned long long
#define ld long double

#define GTX_1060_BLOCKS 1280
#define WARP_SIZE 32 // количество потоков в блоке

/**
 * Запуск по всем блокам. После выполнения функции в per_blocks_sum лежат
 * сумма для каждого блока, и тогда эти значения суммируются в одно.
 * @param per_blocks_sum массив с локальной суммой по всем потокам каждого блока
 * @param iterations количество итераций на поток.
 */
__global__ void kernel(ull *per_blocks_sum, ull iterations) {
    __shared__ ull per_block_sum[WARP_SIZE];
    ull index = threadIdx.x + blockIdx.x * blockDim.x;

    hiprandState_t rng;
    hiprand_init(clock64(), index, 0, &rng);

    per_block_sum[threadIdx.x] = 0;

    for (int i = 0; i < iterations; i++) {
        double x = hiprand_uniform(&rng); // x в [0,1]
        double y = hiprand_uniform(&rng); // y в [0,1]
        per_block_sum[threadIdx.x] += 1 - int(x * x + y * y);
    }

    if (threadIdx.x == 0) {
        per_blocks_sum[blockIdx.x] = 0;
        for (int i = 0; i < WARP_SIZE; i++) {
            per_blocks_sum[blockIdx.x] += per_block_sum[i];
        }
    }
}

__host__ ld monteCarloCPU(ull N) {
    double x,y;
    ld sum = 0;
    for(int i = 0; i < N; i++){
        x = (double) rand()/RAND_MAX;
        y = (double) rand()/RAND_MAX;
        if(x*x + y*y <= 1) sum += 1.0;
    }
    return sum * 4.0 / (ld)(N);
}

__host__ ld monteCarloGPU(ull N) {
    ull iterations;
    size_t size = N * sizeof(ull);

    ull *sums_per_blocks = nullptr;

    hipMalloc(&sums_per_blocks, size);

    iterations = N / (GTX_1060_BLOCKS * WARP_SIZE);
    if (iterations == 0) {
        iterations = 1;
        kernel<<<N, 1>>>(sums_per_blocks, iterations);
    }
    else {
        kernel<<<GTX_1060_BLOCKS, WARP_SIZE>>>(sums_per_blocks, iterations);
    }
    hipDeviceSynchronize();

    ull *host_sums_per_blocks = (ull *) malloc(size);
    hipMemcpy(host_sums_per_blocks, sums_per_blocks, size, hipMemcpyDeviceToHost);

    double sum = 0;
    double sum_iterations  = GTX_1060_BLOCKS;
    if(iterations == 1) {
        sum_iterations = N;
    }

    for (int i = 0; i < sum_iterations; i++) {
        sum += host_sums_per_blocks[i];
    }
    double divizor = iterations == 1 ? N : GTX_1060_BLOCKS * WARP_SIZE * iterations;

    free(host_sums_per_blocks);
    hipFree(sums_per_blocks);

    return sum * 4 / divizor;
}

int main() {
    unsigned long long n = 1e8;
//    scanf("%llu", &n);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    ld pi = monteCarloGPU(n);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    printf("GPU Pi:: %Lf\n", pi);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time consumed for monteCarloGPU :: %3.1f ms \n", milliseconds);

    hipEventRecord(start, 0);
    pi = monteCarloCPU(n);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    printf("CPU Pi:: %Lf\n", pi);

    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time consumed for monteCarloCPU :: %3.1f ms \n", milliseconds);
    return 0;
}